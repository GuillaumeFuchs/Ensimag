# include "option.h"
# include "asian.h"
# include <pnl/pnl_mathtools.h>
#include <pnl/pnl_vector.h>

#include "asian.cuh"

/*!
 * \file asian.cpp
 * \brief Implémentation de la classe fille d'Option: Asian 
 * \author equipe 11
 */

Asian :: Asian() : Option() {
  Strike_ = 0.;
}

Asian::Asian(Parser &pars) : Option(pars){
  Strike_ = pars.getDouble("strike");
}

Asian :: ~Asian(){
}

double Asian :: get_Strike() {
  return Strike_;
}

void Asian :: set_Strike(double Strike){
  Strike_=Strike;
}

double Asian :: payoff (const PnlMat *path) {
  double sum;
  //Vecteur pour mette les valeurs des S_{ti}
  //Dimension D=1 donc path ne contient qu'une seule ligne (indice 0)
  PnlVect* final = pnl_vect_create(TimeSteps_+1);
  pnl_mat_get_row(final ,path, 0);
  //Calcul d'une option asiatique discrète
  sum = (1/(double)(TimeSteps_))*pnl_vect_sum(final) - Strike_;
  pnl_vect_free(&final);
  return MAX(sum, 0);
}

void Asian::price_mc(
	double &prix,
	int nBlocks,
	int nThreads,
	int N,
	int samples,
	float* d_path) 
{
	//Compute price
	float* d_per_block_results_price;
	hipMalloc((float**)&d_per_block_results_price, nBlocks*sizeof(float));

	mc_asian<<<nBlocks, nThreads, nBlocks*sizeof(float)>>>(N, size_, samples, (float)Strike_, d_path, d_per_block_results_price);
	hipDeviceSynchronize();

	float* per_block_results_price = (float*)malloc(nBlocks*sizeof(float));
	hipMemcpy(per_block_results_price, d_per_block_results_price, nBlocks*sizeof(float), hipMemcpyDeviceToHost);

	prix = 0.0;
	for (int i = 0; i < nBlocks; i++){
		prix += per_block_results_price[i];
	}
	hipFree(d_per_block_results_price);
}
